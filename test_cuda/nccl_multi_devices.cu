#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <thread>
#include "hip/hip_runtime.h"
#include "nccl.h"

#include "mpi.h"
#include <unistd.h>
#include <stdint.h>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

static uint64_t getHostHash(const char* string) {
  // Based on DJB2, result = result * 33 + char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) + string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

void print_result(float* buff, int size) {
  float* h_buff = new float[size];
  CUDACHECK(hipMemcpy(h_buff, buff, sizeof(float) * size, hipMemcpyDeviceToHost));
  std::cout << "buff size: " << size << std::endl;
  for (int i=0; i<size; i++) {
    std::cout << h_buff[i] << " ";
  }
  std::cout << std::endl;
  delete[] h_buff;
}

void test_nccl_multi_device() {
  ncclComm_t comms[4];


  //managing 4 devices
  int nDev = 4;
  int size = 32*1024*1024;
  int devs[4] = { 0, 1, 2, 3 };


  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  // host sendbuff
  float* h_sendbuff = new float[size];
  std::fill_n(h_sendbuff, size, 1);




  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    // CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemcpy(sendbuff[i], h_sendbuff, size * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  std::cout << "sendbuff: " << std::endl;
  print_result(sendbuff[0], 10);
  std::cout << "recvbuff: " << std::endl;
  print_result(recvbuff[0], 10);

  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));


   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i)
    // NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
    //     comms[i], s[i]));
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)sendbuff[i], size, ncclFloat, ncclSum,
        comms[i], s[i]));
  NCCLCHECK(ncclGroupEnd());

  for (int i=0; i<nDev; i++) {
    std::cout << "after allreduce device id: " << i << " recvbuff: " << std::endl;
    print_result(recvbuff[0], 10);
  }
  for (int i=0; i<nDev; i++) {
    std::cout << "after allreduce device id: " << i << " sendbuff: " << std::endl;
    print_result(sendbuff[0], 10);
  }
  


  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }


  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);
  
  printf("test multi device Success \n");
}

void test_nccl_multi_process() {
  int size = 32*1024*1024;
  int myRank, nRanks, localRank = 0;


  //initializing MPI
  // MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  std::cout << "myRank: " << myRank << " nRanks: " << nRanks << std::endl;

  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);

  std::cout << "hostname: " << hostname << std::endl;
  for (int i=0; i<nRanks; i++) {
    std::cout << "rank: " << i << " hash: " << hostHashs[i] << std::endl;
  }

  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  for (int i=0; i<nRanks; i++) {
    std::cout << "rank: " << i << " hash: " << hostHashs[i] << std::endl;
  }


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));


  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
        comm, s));


  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);

}


__global__ void do_average(float* data, int ndev, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size) return;
    data[tid] /= ndev;
}


void nccl_comm_multi_thread(int myRank, int nRanks, ncclUniqueId& id) {
  // ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff;
  hipStream_t s;
  int size = 32;

  float* h_sendbuff = new float[size];
  std::fill_n(h_sendbuff, size, 1 + myRank);


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(myRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMemcpy(sendbuff, h_sendbuff, size * sizeof(float), hipMemcpyHostToDevice));
  CUDACHECK(hipStreamCreate(&s));


  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)sendbuff, size, ncclFloat, ncclSum,
        comm, s));


  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  do_average<<<1, 256>>>(sendbuff, nRanks, size);

  std::cout << "[mutip thread Rank " << myRank << "] Success \n";
  if (myRank == 0) print_result(sendbuff, size);

  //free device buffers
  CUDACHECK(hipFree(sendbuff));

  delete[] h_sendbuff;


  //finalizing NCCL
  ncclCommDestroy(comm);
}

void test_nccl_multi_thread() {
  int ndev = 8;
  ncclUniqueId id;
  ncclGetUniqueId(&id);

  std::thread threads[ndev];
    for (int i=0; i<ndev; i++) {
        threads[i] = std::thread(nccl_comm_multi_thread, i, ndev, std::ref(id));
    }
    
    for (int i=0; i<ndev; i++) {
        threads[i].join();
    }
}

int main(int argc, char* argv[])
{
  // test_nccl_multi_device();
  // test_nccl_multi_process();
  test_nccl_multi_thread();

  
  return 0;
}