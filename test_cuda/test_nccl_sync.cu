#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <mutex>
#include <condition_variable>

#include <map>
#include <assert.h>
#include <string>
#include <thread>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"
#include "nccl.h"


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


void print_result(float* buff, int size) {
  float* h_buff = new float[size];
  CUDACHECK(hipMemcpy(h_buff, buff, sizeof(float) * size, hipMemcpyDeviceToHost));
  std::cout << "buff size: " << size << std::endl;
  for (int i=0; i<size; i++) {
    std::cout << h_buff[i] << " ";
  }
  std::cout << std::endl;
  delete[] h_buff;
}

template<class T>
class GlobalSharedRank {
 public:
  T Register(const std::string &key, int ndev) {
    std::lock_guard<std::mutex> lock(mutex_);
    auto it = registry_.find(key);
    if (it != registry_.end()) {
      T* tmpT = it->second;
      *tmpT = (*tmpT == ndev - 1) ? 0 : *tmpT + 1;
      return *tmpT;
    }
    T *newT = new T(0);
    registry_[key] = newT;
    return *newT;
  }
  ~GlobalSharedRank() {
    for (auto it = registry_.begin(); it != registry_.end(); it++) {
      T *ptr = it->second;
      delete ptr;
    }
  }
 private:
  std::mutex mutex_;
  std::map<std::string, T*> registry_;
};

__global__ void do_average(float* data, int ndev, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size) return;
    data[tid] /= ndev;
}

class GlobalShared {
private:
    std::vector<float*> tensors;
    // float** tensors;
    int tensor_size=-1;
    bool* ready_flags;
    ncclComm_t* comms = nullptr;
    hipStream_t* s = nullptr;
    int ndev;
    bool mean_ready;
    std::mutex mutex_;

public:
    GlobalShared(int ndev, int size):ndev(ndev), tensor_size(size) {
        comms = new ncclComm_t[ndev];
        int devs[ndev];
        s = new hipStream_t[ndev];
        ready_flags = new bool[ndev];
        // tensors = new float*[ndev];
        tensors = std::vector<float*>(ndev, nullptr);

        for (int i=0; i<ndev; i++) {
            devs[i]=i;
            CUDACHECK(hipSetDevice(i));
            CUDACHECK(hipStreamCreate(s+i));
        }
        NCCLCHECK(ncclCommInitAll(comms, ndev, devs));

        // for (int i=0; i<ndev; i++)
        //     std::cout << "cudastream " << i << " : " << s+i << std::endl;
    }

    GlobalShared(const GlobalShared& other) {
        *this = other;
    }

    const GlobalShared& operator=(const GlobalShared& other) {
        *this = other;
        return *this;
    }

    ~GlobalShared() {
        std::cout << "~global shared called" << std::endl;
        for(int i = 0; i < ndev; ++i) {
            ncclCommDestroy(comms[i]);
        }
        delete[] comms;
        delete[] s;
        // delete[] tensors;
        delete[] ready_flags;
    }

    void set_tensor_ptr(float* tensor, int rank) {
        std::lock_guard<std::mutex> lck(mutex_);
        tensors[rank] = tensor;
        ready_flags[rank] = true;
    }

    void get_mean(int rank) {
        while(!MeanReady()) {}
        ready_flags[rank] = false;
        resetMeanReady();
    }

    bool MeanReady() {
        std::lock_guard<std::mutex> lck(mutex_);
        if (mean_ready) return true;
        for (int i=0; i<ndev; i++) {
            if (!ready_flags[i])  return false;
        }

        NCCLCHECK(ncclGroupStart());
        for (int i = 0; i < ndev; ++i) {
            float* buffi = tensors[i];
            NCCLCHECK(ncclAllReduce((const void*)buffi, (void*)buffi, tensor_size, ncclFloat, ncclSum, comms[i], s[i]));
        }
        NCCLCHECK(ncclGroupEnd());

        for (int i = 0; i < ndev; ++i) {
            CUDACHECK(hipSetDevice(i));
            std::cout << "sync " << i << ": " << &(s[i]) << std::endl;
            CUDACHECK(hipStreamSynchronize(s[i]));
        }
    
        // // do average
        // for (int i=0; i<ndev; i++) {
        //     do_average<<<1, 256>>>(tensors[i], ndev, tensor_size);
        // }

        mean_ready = true;
        return true;
    }

    void resetMeanReady() {
        for (int i=0; i<ndev; i++) {
            if (ready_flags[i]) return;
        }
        mean_ready = false;
    }

};

template <typename T>
class singleton {
private:
    singleton(){}
    ~singleton(){}
public:
    static T* getInstance(int ndev, int size) {
        static T m_instance = T(ndev, size);
        return &m_instance;
    }
};


static GlobalSharedRank<int> global_shared_rank;


void sync_func(int dev_id, int ndev) {
    int size = 32;
    float* buff;
    // host sendbuff
    float* h_sendbuff = new float[size];
    std::fill_n(h_sendbuff, size, 1 + dev_id);

    CUDACHECK(hipSetDevice(dev_id));
    CUDACHECK(hipMalloc(&buff, size * sizeof(float)));
    CUDACHECK(hipMemcpy(buff, h_sendbuff, size * sizeof(float), hipMemcpyHostToDevice));
    
    static GlobalShared* gs = singleton<GlobalShared>::getInstance(ndev, size);
    (*gs).set_tensor_ptr(buff, dev_id);

    (*gs).get_mean(dev_id);
    if (dev_id == 0) {
        std::cout << dev_id << "/" << ndev << " allreduce:\n";
        print_result(buff, 10);
    }
}

void sync_func_single_thread(int ndev) {
    int size = 32;
    std::vector<float*> h_sendbuffs = std::vector<float*>(ndev, nullptr);
    std::vector<float*> buffs = std::vector<float*>(ndev, nullptr);
    static GlobalShared* gs = singleton<GlobalShared>::getInstance(ndev, size);
    for (int dev_id=0; dev_id<ndev; dev_id++) {
        h_sendbuffs[dev_id] = new float[size];
        std::fill_n(h_sendbuffs[dev_id], size, 1 + dev_id);

        CUDACHECK(hipSetDevice(dev_id));
        CUDACHECK(hipMalloc(&buffs[dev_id], size * sizeof(float)));
        CUDACHECK(hipMemcpy(buffs[dev_id], h_sendbuffs[dev_id], size * sizeof(float), hipMemcpyHostToDevice));
        (*gs).set_tensor_ptr(buffs[dev_id], dev_id);
    }

    int dev_id = 0;
    (*gs).get_mean(dev_id);
    if (dev_id == 0) {
        std::cout << dev_id << "/" << ndev << " allreduce:\n";
        print_result(buffs[dev_id], 10);
    }


}

void test_sync(int ndev) {
    std::thread threads[ndev];
    for (int i=0; i<ndev; i++) {
        threads[i] = std::thread(sync_func, i, ndev);
    }
    
    for (int i=0; i<ndev; i++) {
        threads[i].join();
    }
}


void test_ptr_ptr() {
    float** tensors;
    bool* flags;
    int ndev =2;
    int size = 32;
    tensors = new float*[ndev];
    flags = new bool[ndev];

    for (int i=0; i<ndev; i++) {
        float* tmp = new float[size];
        std::cout << "tensors " << i << " address: " << tmp << std::endl;
        std::fill_n(tmp, size, i+1);
        tensors[i]  = tmp;
        flags[i] = true;
    }

    std::cout << "tensors address: " << tensors << std::endl;
    for (int i=0; i<ndev; i++) {
        std::cout << "tensors " << i << " address: " << tensors[i] << std::endl;
    }

    for (int i=0; i<ndev; i++) {
        delete[] tensors[i];
    }
    delete[] tensors;
    delete[] flags;

}

void test_do_average() {
    int size = 32;
    int ndev = 2;
    int dev_id = 1;
    float* buff;
    // host sendbuff
    float* h_sendbuff = new float[size];
    std::fill_n(h_sendbuff, size, 1 + dev_id);

    CUDACHECK(hipSetDevice(dev_id));
    CUDACHECK(hipMalloc(&buff, size * sizeof(float)));
    CUDACHECK(hipMemcpy(buff, h_sendbuff, size * sizeof(float), hipMemcpyHostToDevice));
    do_average<<<1, 256>>>(buff, ndev, size);
    print_result(buff, 10);
}

int main(int argc, char* argv[])
{
    // test_sync(2);
    sync_func_single_thread(2);
    // test_ptr_ptr();
    // test_do_average();
  
  return 0;
}