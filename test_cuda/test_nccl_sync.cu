#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <mutex>
#include <condition_variable>

#include <map>
#include <assert.h>
#include <string>
#include <thread>
#include <string>

#include "hip/hip_runtime.h"
#include "nccl.h"


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


void print_result(float* buff, int size) {
  float* h_buff = new float[size];
  CUDACHECK(hipMemcpy(h_buff, buff, sizeof(float) * size, hipMemcpyDeviceToHost));
  std::cout << "buff size: " << size << std::endl;
  for (int i=0; i<size; i++) {
    std::cout << h_buff[i] << " ";
  }
  std::cout << std::endl;
  delete[] h_buff;
}

template<class T>
class GlobalSharedRank {
 public:
  T Register(const std::string &key, int ndev) {
    std::lock_guard<std::mutex> lock(mutex_);
    auto it = registry_.find(key);
    if (it != registry_.end()) {
      T* tmpT = it->second;
      *tmpT = (*tmpT == ndev - 1) ? 0 : *tmpT + 1;
      return *tmpT;
    }
    T *newT = new T(0);
    registry_[key] = newT;
    return *newT;
  }
  ~GlobalSharedRank() {
    for (auto it = registry_.begin(); it != registry_.end(); it++) {
      T *ptr = it->second;
      delete ptr;
    }
  }
 private:
  std::mutex mutex_;
  std::map<std::string, T*> registry_;
};

__global__ void do_average(float* data, int ndev, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > size) return;
    data[tid] /= ndev;
}

class GlobalShared {
private:
    float** tensors;
    int tensor_size=-1;
    bool* ready_flags;
    ncclComm_t* comms = nullptr;
    hipStream_t* s = nullptr;
    int ndev;
    bool mean_ready;
    std::mutex mutex_;

public:
    GlobalShared(int ndev, int size):ndev(ndev), tensor_size(size) {
        comms = new ncclComm_t[ndev];
        int devs[ndev];
        s = new hipStream_t[ndev];
        ready_flags = new bool[ndev];
        tensors = new float*[ndev];

        for (int i=0; i<ndev; i++) {
            devs[i]=i;
            CUDACHECK(hipSetDevice(i));
            CUDACHECK(hipStreamCreate(s+i));
        }
        NCCLCHECK(ncclCommInitAll(comms, ndev, devs));

        for (int i=0; i<ndev; i++)
            std::cout << "cudastream " << i << " : " << s+i << std::endl;
    }

    GlobalShared(const GlobalShared& other) {
        *this = other;
    }

    const GlobalShared& operator=(const GlobalShared& other) {
        *this = other;
        return *this;
    }

    ~GlobalShared() {
        std::cout << "~global shared called" << std::endl;
        for(int i = 0; i < ndev; ++i) {
            ncclCommDestroy(comms[i]);
        }
        delete[] comms;
        delete[] s;
        delete[] tensors;
        delete[] ready_flags;
    }

    void set_tensor_ptr(float* tensor, int rank) {
        std::lock_guard<std::mutex> lck(mutex_);
        tensors[rank] = tensor;
        ready_flags[rank] = true;
    }

    void get_mean(int rank) {
        std::unique_lock<std::mutex> lck(mutex_);
        while(!MeanReady()) {}
        ready_flags[rank] = false;
        resetMeanReady();
    }

    bool MeanReady() {
        if (mean_ready) return true;
        for (int i=0; i<ndev; i++) {
            bool tmp = ready_flags[i];
            if (!tmp)  return false;
            // if (!ready_flags[key][i])  return false;
        }

        NCCLCHECK(ncclGroupStart());
        for (int i = 0; i < ndev; ++i) {
            float* buffi = tensors[i];
            NCCLCHECK(ncclAllReduce((const void*)buffi, (void*)buffi, tensor_size, ncclFloat, ncclSum, comms[i], s[i]));
            // NCCLCHECK(ncclAllReduce((const void*)buff[i], (void*)buff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
        }
        NCCLCHECK(ncclGroupEnd());

        for (int i = 0; i < ndev; ++i) {
            CUDACHECK(hipSetDevice(i));
            std::cout << "sync " << i << ": " << &(s[i]) << std::endl;
            CUDACHECK(hipStreamSynchronize(s[i]));
        }
    
        // do average
        for (int i=0; i<ndev; i++) do_average<<<1, 256>>>(tensors[i], ndev, tensor_size);

        mean_ready = true;
        return true;
    }

    void resetMeanReady() {
        for (int i=0; i<ndev; i++) {
            if (ready_flags[i]) return;
        }
        mean_ready = false;
    }

};

template <typename T>
class singleton {
private:
    singleton(){}
    ~singleton(){}
public:
    static T* getInstance(int ndev, int size) {
        static T m_instance = T(ndev, size);
        return &m_instance;
    }
};


static GlobalSharedRank<int> global_shared_rank;


void sync_func(int dev_id, int ndev) {
    int size = 32;
    float* buff;
    // host sendbuff
    float* h_sendbuff = new float[size];
    std::fill_n(h_sendbuff, size, 1 + dev_id);

    CUDACHECK(hipSetDevice(dev_id));
    CUDACHECK(hipMalloc(&buff, size * sizeof(float)));
    CUDACHECK(hipMemcpy(buff, h_sendbuff, size * sizeof(float), hipMemcpyHostToDevice));
    
    static GlobalShared* gs = singleton<GlobalShared>::getInstance(ndev, size);
    (*gs).set_tensor_ptr(buff, dev_id);

    (*gs).get_mean(dev_id);

    std::cout << dev_id << "/" << ndev << " allreduce:\n";
    print_result(buff, 10);
}


void test_sync() {
    int ndev = 1;
    std::thread threads[ndev];
    for (int i=0; i<ndev; i++) {
        threads[i] = std::thread(sync_func, i, ndev);
    }
    
    for (int i=0; i<ndev; i++) {
        threads[i].join();
    }
}

int main(int argc, char* argv[])
{
    test_sync();
  
  return 0;
}