#include <vector>
#include <thread>
#include <algorithm>
#include <iostream>

#include "hip/hip_runtime.h"
#include "nccl.h"


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)



__global__ void do_average(float* data, int ndev, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size) return;
    data[tid] /= ndev;
}


void print_result(float* buff, int size) {
  float* h_buff = new float[size];
  CUDACHECK(hipMemcpy(h_buff, buff, sizeof(float) * size, hipMemcpyDeviceToHost));
  std::cout << "buff size: " << size << std::endl;
  for (int i=0; i<size; i++) {
    std::cout << h_buff[i] << " ";
  }
  std::cout << std::endl;
  delete[] h_buff;
}

template <typename T>
class singleton {
private:
    singleton(){}
    ~singleton(){}
public:
    static T& getInstance(int ndev) {
        static T m_instance = T(ndev);
        return m_instance;
    }
};

class GlobalShared {
private:
    int ndev;
    ncclUniqueId uid;
    ncclComm_t* comms;
    hipStream_t* streams;
    std::vector<bool> inited;
    int NUM_THREADS = 256;

public:
    GlobalShared(int ndev): ndev(ndev) {
            ncclGetUniqueId(&uid);
            inited = std::vector<bool>(ndev, false);
            comms = new ncclComm_t[ndev];
            streams = new hipStream_t[ndev];
    }

    ~GlobalShared() {
        for(int i=0; i < ndev; i++) {
            if (inited[i]) ncclCommDestroy(comms[i]);
        }
        delete comms;
        delete streams;
    }

    void init(int rank) {
        if (!inited[rank]) {
            CUDACHECK(hipSetDevice(rank));
            CUDACHECK(hipStreamCreate(&streams[rank]));
            NCCLCHECK(ncclCommInitRank(&comms[rank], ndev, uid, rank));
            inited[rank] = true;
        }
    }

    void reduce(float* buff, int size, int rank) {
        NCCLCHECK(ncclAllReduce((const void*)buff, (void*)buff, size, ncclFloat, ncclSum, comms[rank], streams[rank]));
        CUDACHECK(hipStreamSynchronize(streams[rank]));

        do_average<<<(size+NUM_THREADS-1)/NUM_THREADS, NUM_THREADS>>>(buff, ndev, size);
    }

};

void bn_forward(int rank, int ndev) {

    int repeat = 10;
    std::vector<int> arr(10, 0);
    for (int i=0; i < repeat; i++) arr[i] = i;

    // std::random_device rd;
    // std::mt19937 g(rd());
    // std::shuffle(arr.begin(), arr.end(), g);
    // for (int i=0; i<repeat; i++) std::cout << arr[i] << " ";
    // std::cout << std::endl;

    for (int i = 0; i < repeat; i++) {
    // for (auto& i: arr) {

        static GlobalShared gs = singleton<GlobalShared>::getInstance(ndev);
        gs.init(rank);
        
        int size = 32 * (i+1);
        // if (rank == 0) {
            std::cout << rank << " all reduce i: " << i << " size: " << size << std::endl;
        // }
        
        float *buff, *h_buff;
        h_buff = new float[size];
        std::fill_n(h_buff, size, (1+rank) + i * ndev);

        CUDACHECK(hipSetDevice(rank));
        CUDACHECK(hipMalloc(&buff, size * sizeof(float)));
        CUDACHECK(hipMemcpy(buff, h_buff, size * sizeof(float), hipMemcpyHostToDevice));
        
        gs.reduce(buff, size, rank);

        if (rank == 0) {
            print_result(buff, 10);
        }

        CUDACHECK(hipFree(buff));

        delete[] h_buff;
    }
}

void test_sync_bn() {
    int ndev = 2;
    std::thread threads[ndev];
    for (int i=0; i<ndev; i++) {
        threads[i] = std::thread(bn_forward, i, ndev);
    }
    for (int i=0; i<ndev; i++) {
        threads[i].join();
    }
}

int main() {
    test_sync_bn();

    return 0;
}